
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 1000000
#define B 1024

__global__ void prescan( float *g_idata, float *INCR, int n);

void scanCPU(float *f_out, float *f_in, int i_n);

double myDiffTime(struct timeval &start, struct timeval &end)
{
        double d_start, d_end;
        d_start = (double)(start.tv_sec + start.tv_usec/1000000.0);
        d_end = (double)(end.tv_sec + end.tv_usec/1000000.0);
        return (d_end - d_start);
}

int main(int argc, char **argv)
{
        float a[N], c[N], g[N];
	
        timeval start, end;
	
        float *dev_a, *dev_g, INCRR[B];
        float size = N*sizeof(float);

//	cudaMallocManaged(&a, N*sizeof(float));
        double d_gpuTime, d_cpuTime;	           
	hipHostAlloc(&dev_a, size, hipHostMallocDefault);
	hipHostAlloc(&dev_g, size, hipHostMallocDefault);

	dev_a = (float *)malloc(size);
        dev_g = (float *)malloc(size);
	hipMalloc((void **)&dev_a, size);
        hipMalloc((void **)&dev_g, size);
	for (int i = 0; i < N; i++)
        {
       
//	a[i] = (float)(rand() % 1000000)/1000.0;
	a[i] = i+1;
        }
	int BATCH = (N/B);
	gettimeofday(&start,NULL);

        hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice); 
        prescan <<< BATCH, B>>> ( dev_a ,INCRR , N); 
	hipDeviceSynchronize();
        hipMemcpy(g, dev_g, size, hipMemcpyDeviceToHost);
        gettimeofday(&end, NULL);
	
        d_gpuTime = myDiffTime(start, end);

        gettimeofday(&start, NULL);
        scanCPU(c, a, N);

        gettimeofday(&end, NULL);
        d_cpuTime = myDiffTime(start, end);


        hipFree(dev_a); hipFree(dev_g);

//      for (int i = 0; i < N; i++)
 //     {
  //            printf("c[%i] = %0.3f, g[%i] = %0.3f\n", i, c[i], i, g[i]);
   //  }
        printf("GPU Time for scan size %i: %f\n", N, d_gpuTime);
        printf("CPU Time for scan size %i: %f\n", N, d_cpuTime);
}

__global__ void prescan( float *g_idata, float *INCR, int n)
{
        extern  __shared__  float temp[], g_odata[], SUMS[];
	// allocated on invocation 
	int thid = threadIdx.x + (blockIdx.x * blockDim.x); 
	int offset = 1; 
    	temp[2*thid]   = g_idata[2*thid]; 
	// load input into shared memory 
    	temp[2*thid+1] = g_idata[2*thid+1]; 
	for (int d = B>>1; d > 0; d >>= 1) 
	// build sum in place up the tree 
    	{ 
        	__syncthreads(); 
		if (thid < d)    
        	{ 
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
		    	temp[bi] += temp[ai];   
		     
  		} 
        	offset *= 2; 
	for(int d =B>>1; d>0; d>>=1){
	//build up sums
	__syncthreads();
	if (thid < d)
              {
                       int ai = offset*(2*thid+1)-1;
                       int bi = offset*(2*thid+2)-1;
			}

    	} 

	if (SUMS && thid == 0) 
	{ 
		SUMS[B] = temp[2*B-1];
		temp[n - 1] = 0; 
	} 

	// clear the last element 
	for (int d = 1; d < n; d *= 2) 
	// traverse down tree & build scan 
    	{ 
        	offset >>= 1; 
        	__syncthreads(); 
		if (thid < d) 
        	{ 
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
			float t   = SUMS[ai]; 
            		SUMS[ai]  = SUMS[bi]; 
           		SUMS[bi] += INCR[bi]; 
      	} 
    	} 
    	__syncthreads(); 
    	temp[2*thid]   = INCR[2*thid]; 
	// write results to device memory 
    //	g_odata[2*thid+1] = temp[2*thid+1]
//	g_odata[2*thid+1] = INCR[thid];  
}
}
void scanCPU(float *f_out, float *f_in, int i_n)
{
        f_out[0] = 0;
        for (int i = 1; i < i_n; i++)
                f_out[i] = f_out[i-1] + f_in[i-1];

}


